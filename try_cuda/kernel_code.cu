#include<hip/hip_runtime.h>
#include<hipsolver.h>
#include<stdio.h>


void diagonalize_symmetric_matrix(double *h_A, int N, double *h_evec, double *h_eval)
{
    double *d_A, *d_W; // Device memory for matrix and eigenvalues
    hipError_t cudaStat;
    hipsolverStatus_t cusolverStat;
    
    // Allocate memory on the device
    cudaStat = hipMalloc((void**)&d_A, N * N * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed!\n");
        return;
    }
    printf("Taseng checking.....\n");
    cudaStat = hipMalloc((void**)&d_W, N * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed!\n");
        hipFree(d_A);
        return;
    }

    // Copy matrix A to device
    cudaStat = hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy failed!\n");
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Create cuSOLVER handle
    hipsolverHandle_t cusolverH;
    cusolverStat = hipsolverDnCreate(&cusolverH);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "cuSOLVER handle creation failed!\n");
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Query buffer size
    int Lwork;
    cusolverStat = hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR,
                                               HIPBLAS_FILL_MODE_LOWER, N, d_A, N, d_W, &Lwork);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "Buffer size query failed!\n");
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Allocate workspace
    double *d_work;
    cudaStat = hipMalloc((void**)&d_work, Lwork * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc for workspace failed!\n");
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    int *d_info;
    cudaStat = hipMalloc((void**)&d_info, sizeof(int));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc for info failed!\n");
        hipFree(d_work);
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Compute eigenvalues and eigenvectors
    cusolverStat = hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
                                    N, d_A, N, d_W, d_work, Lwork, d_info);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "Eigen decomposition failed!\n");
        hipFree(d_info);
        hipFree(d_work);
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Copy results back to host

    // Copy back eigenvalues
    cudaStat = hipMemcpy(h_eval, d_W, N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for eigenvalues failed!\n");
    }

    // Copy back eigenvectors
    cudaStat = hipMemcpy(h_evec, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for eigenvectors failed!\n");
    }

    int info;
    cudaStat = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for info failed!\n");
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_W);
    hipFree(d_work);
    hipFree(d_info);
    hipsolverDnDestroy(cusolverH);
}