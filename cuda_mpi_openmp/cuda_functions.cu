#include<stdio.h>
#include<stdlib.h>

#include<hip/hip_runtime.h>
#include<hipsolver.h>

#include<hipblas.h>
#include<hip/hip_complex.h>


void cuda_diagonalize_matrix(double *h_A, int N, double *h_evec, double *h_eval)
{
    double *d_A, *d_W; // Device memory for matrix and eigenvalues
    hipError_t cudaStat;
    hipsolverStatus_t cusolverStat;
    
    // Allocate memory on the device
    cudaStat = hipMalloc((void**)&d_A, N * N * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed!\n");
        return;
    }
    
    cudaStat = hipMalloc((void**)&d_W, N * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed!\n");
        hipFree(d_A);
        return;
    }

    // Copy matrix A to device
    cudaStat = hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy failed!\n");
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Create cuSOLVER handle
    hipsolverHandle_t cusolverH;
    cusolverStat = hipsolverDnCreate(&cusolverH);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "cuSOLVER handle creation failed!\n");
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Query buffer size
    int Lwork;
    cusolverStat = hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR,
                                               HIPBLAS_FILL_MODE_LOWER, N, d_A, N, d_W, &Lwork);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "Buffer size query failed!\n");
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Allocate workspace
    double *d_work;
    cudaStat = hipMalloc((void**)&d_work, Lwork * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc for workspace failed!\n");
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    int *d_info;
    cudaStat = hipMalloc((void**)&d_info, sizeof(int));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc for info failed!\n");
        hipFree(d_work);
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Compute eigenvalues and eigenvectors
    cusolverStat = hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
                                    N, d_A, N, d_W, d_work, Lwork, d_info);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "Eigen decomposition failed!\n");
        hipFree(d_info);
        hipFree(d_work);
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Copy results back to host

    // Copy back eigenvalues
    cudaStat = hipMemcpy(h_eval, d_W, N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for eigenvalues failed!\n");
    }

    // Copy back eigenvectors
    cudaStat = hipMemcpy(h_evec, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for eigenvectors failed!\n");
    }

    int info;
    cudaStat = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for info failed!\n");
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_W);
    hipFree(d_work);
    hipFree(d_info);
    hipsolverDnDestroy(cusolverH);
}


void cuda_complexMatrixMultiply(const hipDoubleComplex *A, const hipDoubleComplex *B, hipDoubleComplex *C, int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hipDoubleComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, n * n * sizeof(hipDoubleComplex));
    hipMalloc((void **)&d_B, n * n * sizeof(hipDoubleComplex));
    hipMalloc((void **)&d_C, n * n * sizeof(hipDoubleComplex));
    
    hipblasSetMatrix(n, n, sizeof(hipDoubleComplex), A, n, d_A, n);
    hipblasSetMatrix(n, n, sizeof(hipDoubleComplex), B, n, d_B, n);
    
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
    
    hipblasGetMatrix(n, n, sizeof(hipDoubleComplex), d_C, n, C, n);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}

void cuda_complexMatrixMultiply_col(const hipDoubleComplex *A, const hipDoubleComplex *B, hipDoubleComplex *C, int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hipDoubleComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, n * n * sizeof(hipDoubleComplex));
    hipMalloc((void **)&d_B, n * 1 * sizeof(hipDoubleComplex));
    hipMalloc((void **)&d_C, n * 1 * sizeof(hipDoubleComplex));
    
    hipblasSetMatrix(n, n, sizeof(hipDoubleComplex), A, n, d_A, n);
    hipblasSetMatrix(n, 1, sizeof(hipDoubleComplex), B, n, d_B, n);
    
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);

    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, 1, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
    
    hipblasGetMatrix(n, 1, sizeof(hipDoubleComplex), d_C, n, C, n);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}